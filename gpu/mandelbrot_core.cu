#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>
#define GETID_MAT_FUNC_PREFIX __device__
#include "../common/mandelbrot.h"


void (*kernel)(complex_t*,int*,int,int,int,int,int,zoominfo_t);

static int tpb = 32;           // Threads per block
static int n_points = 1;       // Points per thread
static int *d_iter;            // Device output buffer for iterations per point
static complex_t *d_mat;       // Device unidimensional buffer for data

__device__
complex_t julia_c = {JULIA_C_REAL, JULIA_C_IMAG};


__global__
void kernel_mandelbrot(complex_t *mat, int *iters, int maxiter, int size, int rows, int cols, int n_points, zoominfo_t zinfo){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    //if(index >= size)
    //    return;
    complex_t z, c;
    double rnew;
    long idxmat = ((long)index) * n_points;
    int i, j, iteration;
    while(idxmat < size && n_points > 0){
        iteration = 0;
        i = idxmat / cols;
        j = idxmat % cols;
        z.r = 0; z.i = 0;
        c.r = ((double)(j - cols/2 + zinfo.start_x))/zinfo.ratio;
        c.i = ((double)(i - rows/2 + zinfo.start_y))/zinfo.ratio;
        while(iteration < maxiter && (z.r*z.r + z.i*z.i) <= 4.0){
            rnew = (z.r * z.r) - (z.i * z.i) + c.r;
            z.i = 2 * (z.r) * (z.i) + c.i;
            z.r = rnew;
            iteration++;
        }
        mat[idxmat].r = z.r;
        mat[idxmat].i = z.i;
        if(iteration == maxiter) iters[idxmat] = -1;  //dentro insieme
        else iters[idxmat] = iteration;
        --n_points;
        ++idxmat;
    }
}


__global__
void kernel_julia(complex_t *mat, int *iters, int maxiter, int size, int rows, int cols, int n_points, zoominfo_t zinfo){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    //if(index >= size)
    //    return;
    complex_t z, c;
    double rnew;
    long idxmat = ((long)index) * n_points;
    int i, j, iteration;
    c = julia_c;
    while(idxmat < size && n_points > 0){
        iteration = 0;
        i = idxmat / cols;
        j = idxmat % cols;
        z.r = ((double)(j - cols/2 + zinfo.start_x))/zinfo.ratio;
        z.i = ((double)(i - rows/2 + zinfo.start_y))/zinfo.ratio;
        while(iteration < maxiter && (z.r*z.r + z.i*z.i) <= 4.0){
            rnew = (z.r * z.r) - (z.i * z.i) + c.r;
            z.i = 2 * (z.r) * (z.i) + c.i;
            z.r = rnew;
            iteration++;
        }
        mat[getid(i,j,cols)].r = z.r;
        mat[getid(i,j,cols)].i = z.i;
        if(iteration == maxiter) iters[idxmat] = -1;  //dentro insieme
        else iters[idxmat] = iteration;
        --n_points;
        ++idxmat;
    }
}

//if(matsize % n_points != 0) ++dim;  // fix es. per 58799
// (matsize+tpb-1)/tpb
void core_computation(){
    int matsize = width * height;
    int dim = matsize / n_points;
    if(matsize % n_points != 0)
        dim++;
    dim3 dimGrid( (dim+tpb-1)/tpb, 1, 1);
    hipError_t err_sync, err_asyn;
    kernel<<<dimGrid, tpb>>>(d_mat, d_iter, max_iter, matsize, height, width, n_points, zoom_info);
    err_sync = hipMemcpy(iterations, d_iter, matsize*sizeof(int), hipMemcpyDeviceToHost);
    err_asyn = hipGetLastError();
    if(err_sync != hipSuccess)
        printf("memcpy error: %s\n", hipGetErrorString(err_sync));
    else if(err_asyn != err_sync)
        printf("async. error: %s\n", hipGetErrorString(err_asyn));
}

int checkCudaParameters(int matsize){
    int n_devices;
    int dimGrid = (matsize+tpb-1) / tpb;
    hipDeviceProp_t prop;
    hipGetDeviceCount(&n_devices);
    if(n_devices == 0){
        printf("no cuda device found\n");
        return 1;
    }
    hipGetDeviceProperties(&prop, 0);
    if(tpb > prop.maxThreadsPerBlock){
        printf("%d threads per block is your limit\n", prop.maxThreadsPerBlock);
        return 1;
    }
    else if(tpb % prop.warpSize != 0){
        printf("block size is not a multiple of the warp size.. \n");
    }
    if(dimGrid > prop.maxGridSize[0]){
        printf("grid dim exceeds your limit, ");
        while(dimGrid > prop.maxGridSize[0]){
            n_points <<= 1;     // n_points*2
            int dim = matsize / n_points;
            if(matsize % n_points != 0) ++dim;
            dimGrid = (dim+tpb-1)/tpb;
        }
        printf("each thread must compute %d points\n", n_points);
    }
    return 0;
}

void core_init(void **args, char *msg){
    int size = width * height;
    if(args[ARG_NUMP] != NULL)
        tpb = (int)strtol((char*)args[ARG_NUMP], NULL, 10);
    if(checkCudaParameters(size)){
        exit(1);
    }
    hipMalloc(&d_mat, size * sizeof(complex_t));
    hipMalloc(&d_iter, size * sizeof(int));
    if(args[ARG_JULY]){
        kernel = kernel_julia;
        if(args[ARG_JULR] != NULL && args[ARG_JULI] != NULL){
            complex_t point;
            point.r = strtod((char*)args[ARG_JULR], NULL);
            point.i = strtod((char*)args[ARG_JULI], NULL);
            hipMemcpyToSymbol(HIP_SYMBOL(julia_c), &point, sizeof(complex_t), 0, hipMemcpyHostToDevice);
        }
    }
    else{
        kernel = kernel_mandelbrot;
    }
    sprintf(msg, "GPU threads per block %d", tpb);
}

void core_exit(void **data){
    hipFree(d_mat);
    hipFree(d_iter);
}
